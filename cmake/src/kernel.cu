#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernel.h"
#include <stdio.h>

__global__ void simpleKernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

extern "C" {
  void runKernel() {
    simpleKernel<<<1, 10>>>();
    hipDeviceSynchronize();
  }
}
